#include "hip/hip_runtime.h"
#include "dominokernel.cuh"
#include <hiprand/hiprand_kernel.h>
#include "../common/hip/hip_runtime_api.h"
#include "stdio.h"

#define PROB_PROPOSE 0.8

__global__ void RotateTilesKernel(hiprandStateMtgp32* d_status, char* tiling, const int N, const int t)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

	// for MTGP indexing
	int id = blockIdx.x * gridDim.y + blockIdx.y;
	float rd = hiprand_uniform(d_status + id);

	if ((i > N - 2) | (j > N / 2 - 2)) { return; }

	if (rd < PROB_PROPOSE) {
		if (tiling[i * (N / 2) + j] == 3) { tiling[i * (N / 2) + j] = 12; }
		else if (tiling[i * (N / 2) + j] == 12) { tiling[i * (N / 2) + j] = 3; }
	}

}


// t is the parity of the tiles being updated, see how this kernel is called in the RandomWalk method
__global__ void UpdateTilesKernel(char* tiling, char* reftiling, const int N, const int t)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if ((i > N - 2) | (j > N / 2 - 2)) { return; }

	tiling[i * (N / 2) + j] = (reftiling[(i - 1) * (N / 2) + j] & 2) / 2
		+ 2 * (reftiling[(i + 1) * (N / 2) + j] & 1)
		+ (reftiling[i * (N / 2) + j - (i + t + 1) % 2] & 8) / 2
		+ 2 * (reftiling[i * (N / 2) + j + (i + t) % 2] & 4);
}


void RotateTiles(dim3 block_size, dim3 grid_size, hiprandStateMtgp32* d_status, char* tiling, const int N, const int t)
{
	RotateTilesKernel <<<grid_size, block_size >>> (d_status, tiling, N, t);
	getLastCudaError("RotateTilesKernel launch failed");
}


void UpdateTiles(dim3 block_size, dim3 grid_size, char* tiling, char* reftiling, const int N, const int t)
{
	UpdateTilesKernel << <grid_size, block_size>> > (tiling, reftiling, N, t);
	getLastCudaError("UpdateTilesKernel launch failed");
}
