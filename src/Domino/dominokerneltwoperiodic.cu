#include "hip/hip_runtime.h"
#include "dominokernel.cuh"
#include <hiprand/hiprand_kernel.h>
#include "../common/hip/hip_runtime_api.h"
#include "stdio.h"

#define PROB_PROPOSE 0.8
#define a 0.7
#define b 1.
// a and b are the weights for the two-periodic weighting. For the uniform weighting, set a and b to 1.

__global__ void RotateTilesKernel(hiprandStateMtgp32* d_status, char* tiling, const int N, const int t)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

	// for MTGP indexing
	int id = blockIdx.x * gridDim.y + blockIdx.y;
	float rd = hiprand_uniform(d_status + id);

	if ((i > N - 2) | (j > N / 2 - 2)) { return; }

	if (rd < PROB_PROPOSE) {
		float threshold = rd / PROB_PROPOSE;
		int e = tiling[i * (N / 2) + j];
		if (e == 3) {
			float wInit = b * b * ((i + t + 1) % 2) + a * a * ((i + t) % 2);
			float wFin = a * a * (i % 2) + b * b * ((i + 1) % 2);
			if (wFin / wInit > threshold) { 
				tiling[i * (N / 2) + j] = 12; 
			}
		}

		else if (e == 12) {
			float wInit = a * a * (i % 2) + b * b * ((i + 1) % 2);
			float wFin = b * b * ((i + t + 1) % 2) + a * a * ((i + t) % 2);
			if (wFin / wInit > threshold) { 
				tiling[i * (N / 2) + j] = 3;
			}
		}
	}

}


// t is the parity of the tiles being updated, see how this kernel is called in the RandomWalk method
__global__ void UpdateTilesKernel(char* tiling, char* reftiling, const int N, const int t)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if ((i > N - 2) | (j > N / 2 - 2)) { return; }

	tiling[i * (N / 2) + j] = (reftiling[(i - 1) * (N / 2) + j] & 2) / 2
		+ 2 * (reftiling[(i + 1) * (N / 2) + j] & 1)
		+ (reftiling[i * (N / 2) + j - (i + t + 1) % 2] & 8) / 2
		+ 2 * (reftiling[i * (N / 2) + j + (i + t) % 2] & 4);
}


void RotateTiles(dim3 block_size, dim3 grid_size, hiprandStateMtgp32* d_status, char* tiling, const int N, const int t)
{
	RotateTilesKernel <<<grid_size, block_size >>> (d_status, tiling, N, t);
	getLastCudaError("RotateTilesKernel launch failed");
}


void UpdateTiles(dim3 block_size, dim3 grid_size, char* tiling, char* reftiling, const int N, const int t)
{
	UpdateTilesKernel << <grid_size, block_size>> > (tiling, reftiling, N, t);
	getLastCudaError("UpdateTilesKernel launch failed");
}
